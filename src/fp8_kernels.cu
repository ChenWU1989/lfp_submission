#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "fp8.h"
#include "hip/hip_runtime.h"
}

__device__ uint8_t fp32_2_fp8(uint32_t src, int exp_width) {
  uint8_t dst;

  uint32_t src_sgn = src & 0x80000000u;
  uint32_t src_exp = src & 0x7F800000u;
  uint32_t src_mts = src & 0x007FFFFFu;

  uint8_t  dst_sgn = (uint8_t)(src_sgn >> 24);
  uint8_t  dst_exp;
  uint8_t  dst_mts;

  int mts_width = 7 - exp_width;
  int fp8_exp_bias = exp_width ? (int)(1 << (exp_width - 1)) - 1 : 0;
  int fp8_exp_max = (int)(1 << exp_width) - 1;
  uint8_t fp8_mts_max = (uint8_t)((1 << mts_width) - 1);
  int dst_exp_value = ((int)(src_exp >> 23)) - 127 + fp8_exp_bias;

  if ((src & 0x7FFFFFFFu) == 0 || src_exp == 0) {                             // signed zero or denormalized number
    dst = (uint8_t)(src >> 24); 
  }
  else {
    if (dst_exp_value > fp8_exp_max) {                                        // overflow to signed "max" value
      dst = ((uint8_t)0x7F) | dst_sgn;
    }
    else if (dst_exp_value <= 0) {                                            // underflow
      if (dst_exp_value < 0 - mts_width) {                                    // Mantissa shifted all the way off
                                                                              // no rounding possibility
        dst_mts = (uint8_t)0u;                                                // Set mantissa to zero
      }
      else {
        src_mts |= 0x00800000u;                                               // Add the hidden leading bit
        dst_mts = (uint8_t)(src_mts >> (24 - mts_width - dst_exp_value));
        if ((src_mts >> (23 - mts_width - dst_exp_value)) & 0x00000001u) {    // Round
          dst_mts += (uint8_t)1u;
        }
      }
      dst = dst_sgn | dst_mts;                                                // Exponent is zero for denormalized number
    }
    else {                                                                    // Normal case
      dst_exp = (uint8_t)(dst_exp_value << mts_width);
      dst_mts = (uint8_t)(src_mts >> (23 - mts_width));
      if (src_mts & (0x00400000u >> mts_width)) {                             // Round  
        if ((dst_exp_value == fp8_exp_max) && (dst_mts == fp8_mts_max))
          dst = ((uint8_t)0x7F) | dst_sgn;
        else
          dst = (dst_sgn | dst_exp | dst_mts) + (uint8_t)1u;
      }
      else {
        dst = (dst_sgn | dst_exp | dst_mts);
      }
    }
  }
  return dst;
}

__device__ uint32_t fp8_2_fp32(uint8_t src, int exp_width)
{
  uint32_t dst;
 
  int mts_width = 7 - exp_width;
  int fp8_exp_bias = exp_width ? (int)(1 << (exp_width - 1)) - 1 : 0;
  uint8_t fp8_mts_max = (uint8_t)((1 << mts_width) - 1);
  
  uint8_t src_sgn = src & 0x80u;
  uint8_t src_exp = src & (0x7F - fp8_mts_max);
  uint8_t src_mts = src & fp8_mts_max;

  uint32_t dst_sgn = ((uint32_t)src_sgn) << 24;
  uint32_t dst_exp;
  uint32_t dst_mts;
  
  int dst_exp_value;

  if ((src & 0x7F) == 0) {                                                    // Signed zero, return signed zero
    dst = ((uint32_t)src) << 24;
  }
  else {
    int e = -1;
    if (src_exp == 0) {                                                       // Denormalized number
                                                                              // Convert to normalized number in fp32
                                                                              // Find the extra to adjust exponent
      do {
        e++;
        src_mts <<= 1;
      } while ((src_mts & (0x01 << mts_width)) == 0);
    }
    else {                                                                    // Normalized number
      e = 0;
    }
    dst_exp_value = ((int)(src_exp >> mts_width)) - fp8_exp_bias + 127 - e;
    dst_exp = (uint32_t)(dst_exp_value << 23);
    dst_mts = ((uint32_t)(src_mts & fp8_mts_max)) << (23 - mts_width);
    dst = dst_sgn | dst_exp | dst_mts;
  }
  return dst;
}

__global__ void fp32_2_fp8_kernel(uint32_t* fp32, uint8_t* fp8, int n, int exp_width)
{
  int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
  if (i < n)
    fp8[i] = fp32_2_fp8(fp32[i], exp_width);
}

__global__ void fp8_2_fp32_kernel(uint8_t* fp8, uint32_t* fp32, int n, int exp_width)
{
  int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
  if (i < n)
    fp32[i] = fp8_2_fp32(fp8[i], exp_width);
}

__device__ float scale_data(float x, float scale)
{
  return x * scale;
}

__global__ void fp8_scale_kernel(float *x, int n, int exp)
{
  int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
  if (i < n)
    x[i] = scale_data(x[i], powf(2, exp));
}

extern "C" void fp8_data_gpu(float *x, int n, int exp, QUANTIZE_TYPE type)
{ 
    fp8_scale_kernel<<<cuda_gridsize(n), BLOCK>>>(x, n, exp); 
    check_error(hipPeekAtLastError());
    
    uint32_t *x_32_bin = (uint32_t*)x;
    uint8_t *x_8_bin;
    hipMalloc(&x_8_bin, sizeof(uint8_t)*n);
    
    int exp_width;
    switch (type) {
      case FP8_M0E7:  exp_width = 7;  break;
      case FP8_M1E6:  exp_width = 6;  break;
      case FP8_M2E5:  exp_width = 5;  break;
      case FP8_M3E4:  exp_width = 4;  break;
      case FP8_M4E3:  exp_width = 3;  break;
      case FP8_M5E2:  exp_width = 2;  break;
      case FP8_M6E1:  exp_width = 1;  break;
      case FP8_M7E0:  exp_width = 0;  break;
      default      :  exp_width = 0;  break;
    }
    
    fp32_2_fp8_kernel<<<cuda_gridsize(n), BLOCK>>>(x_32_bin, x_8_bin, n, exp_width);
    check_error(hipPeekAtLastError());
    fp8_2_fp32_kernel<<<cuda_gridsize(n), BLOCK>>>(x_8_bin, x_32_bin, n, exp_width);
    check_error(hipPeekAtLastError());
    x = (float*)x_32_bin;
    
    hipFree(x_8_bin);
}

extern "C" void fp8_de_data_gpu(float *x, int n, int exp)
{   
    fp8_scale_kernel<<<cuda_gridsize(n), BLOCK>>>(x, n, -exp);
    check_error(hipPeekAtLastError());
}

__global__ void fp8_normalize_kernel(float *x, float *scales, float *biases, int filters, int spatial, size_t n)
{
  int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
  if (i >= n) return;
  int f = (i / spatial) % filters;
  x[i] = scales[f] * x[i] + biases[f];
}

extern "C" void fp8_normalize_gpu(float *x, float *scales, float *biases, int batch, int filters, int spatial)
{
  size_t n = batch * filters * spatial;
  fp8_normalize_kernel<<<cuda_gridsize(n), BLOCK>>>(x, scales, biases, filters, spatial, n);
  check_error(hipPeekAtLastError());
}
