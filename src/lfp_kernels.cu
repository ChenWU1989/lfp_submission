#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "lfp.h"
#include "hip/hip_runtime.h"
}

__device__ uint8_t fp2lfp(uint32_t src, int bit_width, int exp_width) {
  uint8_t dst;

  uint32_t src_sgn = src & 0x80000000u;
  uint32_t src_exp = src & 0x7F800000u;
  uint32_t src_mts = src & 0x007FFFFFu;

  uint8_t  dst_sgn = (uint8_t)(src_sgn >> (32 - bit_width));
  uint8_t  dst_exp;
  uint8_t  dst_mts;

  int mts_width = bit_width - 1 - exp_width;
  int lfp_exp_bias = exp_width ? (int)(1 << (exp_width - 1)) - 1 : 0;
  int lfp_exp_max = (int)(1 << exp_width) - 1;
  uint8_t lfp_mts_max = (uint8_t)((1 << mts_width) - 1);
  int dst_exp_value = ((int)(src_exp >> 23)) - 127 + lfp_exp_bias;

  if ((src & 0x7FFFFFFFu) == 0 || src_exp == 0) {                             // signed zero or denormalized number
    dst = (uint8_t)(src >> (32 - bit_width)); 
  }
  else {
    if (dst_exp_value > lfp_exp_max) {                                        // overflow to signed "max" value
      dst = ((uint8_t)(0x7F >> (8 - bit_width))) | dst_sgn;
    }
    else if (dst_exp_value <= 0) {                                            // underflow
      if (dst_exp_value < 0 - mts_width) {                                    // Mantissa shifted all the way off
                                                                              // no rounding possibility
        dst_mts = (uint8_t)0u;                                                // Set mantissa to zero
      }
      else {
        src_mts |= 0x00800000u;                                               // Add the hidden leading bit
        dst_mts = (uint8_t)(src_mts >> (24 - mts_width - dst_exp_value));
        if ((src_mts >> (23 - mts_width - dst_exp_value)) & 0x00000001u) {    // Round
          dst_mts += (uint8_t)1u;
        }
      }
      dst = dst_sgn | dst_mts;                                                // Exponent is zero for denormalized number
    }
    else {                                                                    // Normal case
      dst_exp = (uint8_t)(dst_exp_value << mts_width);
      dst_mts = (uint8_t)(src_mts >> (23 - mts_width));
      if (src_mts & (0x00400000u >> mts_width)) {                             // Round  
        if ((dst_exp_value == lfp_exp_max) && (dst_mts == lfp_mts_max))
          dst = ((uint8_t)(0x7F >> (8 - bit_width))) | dst_sgn;
        else
          dst = (dst_sgn | dst_exp | dst_mts) + (uint8_t)1u;
      }
      else {
        dst = (dst_sgn | dst_exp | dst_mts);
      }
    }
  }
  return dst;
}

__device__ uint32_t lfp2fp(uint8_t src, int bit_width, int exp_width)
{
  uint32_t dst;
 
  int mts_width = bit_width - 1 - exp_width;
  int lfp_exp_bias = exp_width ? (int)(1 << (exp_width - 1)) - 1 : 0;
  uint8_t lfp_mts_max = (uint8_t)((1 << mts_width) - 1);
  
  uint8_t src_sgn = src & (0x80u >> (8 - bit_width));
  uint8_t src_exp = src & ((0x7F >> (8 - bit_width)) - lfp_mts_max);
  uint8_t src_mts = src & lfp_mts_max;

  uint32_t dst_sgn = ((uint32_t)src_sgn) << (32 - bit_width);
  uint32_t dst_exp;
  uint32_t dst_mts;
  
  int dst_exp_value;

  if ((src & (0x7F >> (8 - bit_width))) == 0) {                               // Signed zero, return signed zero
    dst = ((uint32_t)src) << (32 - bit_width);
  }
  else {
    int e = -1;
    if (src_exp == 0) {                                                       // Denormalized number
                                                                              // Convert to normalized number in fp32
                                                                              // Find the extra to adjust exponent
      do {
        e++;
        src_mts <<= 1;
      } while ((src_mts & (0x01 << mts_width)) == 0);
    }
    else {                                                                    // Normalized number
      e = 0;
    }
    dst_exp_value = ((int)(src_exp >> mts_width)) - lfp_exp_bias + 127 - e;
    dst_exp = (uint32_t)(dst_exp_value << 23);
    dst_mts = ((uint32_t)(src_mts & lfp_mts_max)) << (23 - mts_width);
    dst = dst_sgn | dst_exp | dst_mts;
  }
  return dst;
}

__global__ void fp2lfp_kernel(uint32_t* fp32, uint8_t* lfp, int n, int bit_width, int exp_width)
{
  int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
  if (i < n)
    lfp[i] = fp2lfp(fp32[i], bit_width, exp_width);
}

__global__ void lfp2fp_kernel(uint8_t* lfp, uint32_t* fp32, int n, int bit_width, int exp_width)
{
  int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
  if (i < n)
    fp32[i] = lfp2fp(lfp[i], bit_width, exp_width);
}

__device__ float scale_data(float x, float scale)
{
  return x * scale;
}

__global__ void lfp_scale_kernel(float *x, int n, int exp)
{
  int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
  if (i < n)
    x[i] = scale_data(x[i], powf(2, exp));
}

extern "C" void lfp_data_gpu(float *x, int n, int exp, QUANTIZE_TYPE type)
{ 
    lfp_scale_kernel<<<cuda_gridsize(n), BLOCK>>>(x, n, exp); 
    check_error(hipPeekAtLastError());
    
    uint32_t *x_32_bin = (uint32_t*)x;
    uint8_t *x_8_bin;
    hipMalloc(&x_8_bin, sizeof(uint8_t)*n);
    
    int bit_width;
    int exp_width;

    switch (type) {
      // fp8
      case M0E7:  {
        bit_width = 8;
        exp_width = 7;  
        break;
      }
      case M1E6:  {
        bit_width = 8;
        exp_width = 6;
        break;
      }
      case M2E5:  {
        bit_width = 8;
        exp_width = 5;  
        break;
      }
      case M3E4:  {
        bit_width = 8;
        exp_width = 4;  
        break;
      }
      case M4E3:  {
        bit_width = 8;
        exp_width = 3;  
        break;
      }
      case M5E2:  {
        bit_width = 8;
        exp_width = 2;  
        break;
      }
      case M6E1:  {
        bit_width = 8;
        exp_width = 1;  
        break;
      }
      case M7E0:  {
        bit_width = 8;
        exp_width = 0;  
        break;
      }
      // fp7
      case M0E6:  {
        bit_width = 7;
        exp_width = 6;
        break;
      }
      case M1E5:  {
        bit_width = 7;
        exp_width = 5;  
        break;
      }
      case M2E4:  {
        bit_width = 7;
        exp_width = 4;  
        break;
      }
      case M3E3:  {
        bit_width = 7;
        exp_width = 3;  
        break;
      }
      case M4E2:  {
        bit_width = 7;
        exp_width = 2;  
        break;
      }
      case M5E1:  {
        bit_width = 7;
        exp_width = 1;  
        break;
      }
      case M6E0:  {
        bit_width = 7;
        exp_width = 0;  
        break;
      }
      // fp6
      case M0E5:  {
        bit_width = 6;
        exp_width = 5;
        break;
      }
      case M1E4:  {
        bit_width = 6;
        exp_width = 4;
        break;
      }
      case M2E3:  {
        bit_width = 6;
        exp_width = 3;
        break;
      }
      case M3E2:  {
        bit_width = 6;
        exp_width = 2;
        break;
      }
      case M4E1:  {
        bit_width = 6;
        exp_width = 1;
        break;
      }
      case M5E0:  {
        bit_width = 6;
        exp_width = 0;
        break;
      }
      // fp5
      case M0E4:  {
        bit_width = 5;
        exp_width = 4;
        break;
      }
      case M1E3:  {
        bit_width = 5;
        exp_width = 3;
        break;
      }
      case M2E2:  {
        bit_width = 5;
        exp_width = 2;
        break;
      }
      case M3E1:  {
        bit_width = 5;
        exp_width = 1;
        break;
      }
      case M4E0:  {
        bit_width = 5;
        exp_width = 0;
        break;
      }
      // fp4
      case M0E3:  {
        bit_width = 4;
        exp_width = 3;
        break;
      }
      case M1E2:  {
        bit_width = 4;
        exp_width = 2;
        break;
      }
      case M2E1:  {
        bit_width = 4;
        exp_width = 1;
        break;
      }
      case M3E0:  {
        bit_width = 4;
        exp_width = 0;
        break;
      }
      default      :  {
        bit_width = 8;
        exp_width = 0;  
        break;
      }
    }
    
    fp2lfp_kernel<<<cuda_gridsize(n), BLOCK>>>(x_32_bin, x_8_bin, n, bit_width, exp_width);
    check_error(hipPeekAtLastError());
    lfp2fp_kernel<<<cuda_gridsize(n), BLOCK>>>(x_8_bin, x_32_bin, n, bit_width, exp_width);
    check_error(hipPeekAtLastError());
    x = (float*)x_32_bin;
    
    hipFree(x_8_bin);
}

extern "C" void lfp_de_data_gpu(float *x, int n, int exp)
{   
    lfp_scale_kernel<<<cuda_gridsize(n), BLOCK>>>(x, n, -exp);
    check_error(hipPeekAtLastError());
}

__global__ void lfp_normalize_kernel(float *x, float *scales, float *biases, int filters, int spatial, size_t n)
{
  int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
  if (i >= n) return;
  int f = (i / spatial) % filters;
  x[i] = scales[f] * x[i] + biases[f];
}

extern "C" void lfp_normalize_gpu(float *x, float *scales, float *biases, int batch, int filters, int spatial)
{
  size_t n = batch * filters * spatial;
  lfp_normalize_kernel<<<cuda_gridsize(n), BLOCK>>>(x, scales, biases, filters, spatial, n);
  check_error(hipPeekAtLastError());
}
