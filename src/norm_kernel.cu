#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "hip/hip_runtime.h"
#include "norm.h"
}

__global__ void norm_kernel(float* x, int n, int c, int fmsz, float* mean, float* var)
{
  int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
  int index_c = (i / fmsz) % c;
  if (i < n)
    x[i] = (x[i] - mean[index_c]) / var[index_c];
}

__global__ void denorm_kernel(float* x, int n, int c, int fmsz, float* mean, float* var)
{
  int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
  int index_c = (i / fmsz) % c;
  if (i < n)
    x[i] = x[i] * var[index_c] + mean[index_c];
}

extern "C" void norm_fm_gpu(float* x, int c, int fmsz, float* mean, float* var)
{
  int n = c * fmsz;
  norm_kernel<<<cuda_gridsize(n), BLOCK>>>(x, n, c, fmsz, mean, var);
  check_error(hipPeekAtLastError());
}

extern "C" void denorm_fm_gpu(float* x, int c, int fmsz, float* mean, float* var)
{
  int n = c * fmsz;
  denorm_kernel<<<cuda_gridsize(n), BLOCK>>>(x, n, c, fmsz, mean, var);
  check_error(hipPeekAtLastError());
}

